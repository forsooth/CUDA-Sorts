#include "hip/hip_runtime.h"
#include "bitonic_sort.h"


__global__ void create_bitonic(int *input,  int stride, int length) {
    int tid = (blockIdx.z * gridDim.x * gridDim.y
                + blockIdx.y * gridDim.x
                + blockIdx.x) * blockDim.x + threadIdx.x; 
    int group = tid /stride;
    int order = tid % stride;
    bool even = group % 2 == 0? 1 : 0;
    
    for( int i = stride; i > 0; i = i /2) {
    
        int group1 = order / i;
        int order1 = order % i;
        int index = group * stride * 2 + group1 * i * 2 + order1;
        int index2 = index + i;
        bool less = input[index] < input[index2] ? 1 : 0;
        bool greater = input[index] > input[index2] ? 1 : 0;
        if ((even && greater)||(!even && less)) {
            int  tmp = input[index];
            input[index] = input[index2];
            input[index2] = tmp;
        } 
        
        __syncthreads();
    }
                
        
}


__global__ void bitonic_sort(int *input,  int stride, int length) {
    int tid = (blockIdx.z * gridDim.x * gridDim.y
                + blockIdx.y * gridDim.x
                + blockIdx.x) * blockDim.x + threadIdx.x;   
    int group = tid /stride;
    int order = tid % stride;
    
    for( int i = stride; i > 0; i = i /2) {
    
        int group1 = order / i;
        int order1 = order % i;
        int index = group * stride * 2 + group1 * i * 2 + order1;
        int index2 = index + i;
        if (input[index] < input[index2]) {
            int  tmp = input[index];
            input[index] = input[index2];
            input[index2] = tmp;        
        
        }
 
        __syncthreads();
    }
                        
        
}


void bitonic_sort(Data *data) {

    int threads;
    Grid grid;
    int tmplength = floor(pow(2, ceil(log2((double)(data->length))) - 1));
    int length = data->length * sizeof(int);
    cal_grid(&grid, &threads, tmplength, 1);

    dim3 blocks(grid.blockx, grid.blocky, grid.blockz);
    clock_t begin, end;
    double time_spent;
        
   
    int *input;
    hipMalloc((void**)&input, length);
    hipMemcpy(input, data->intarray, length, hipMemcpyHostToDevice);
    begin = clock();
    int i = 1;
    for (; i < data->length/2; i *= 2) {
        create_bitonic<<<blocks, threads>>>(input, i, data->length);
    }
    create_bitonic<<<blocks, threads>>>(input, i, data->length);
    hipMemcpy(data->intarray, input, length, hipMemcpyDeviceToHost);
    end = clock();
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    fprintf(stdout, "Parallel Merge sort time: %f\n", time_spent);
    hipFree(input);

}
