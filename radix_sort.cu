#include "hip/hip_runtime.h"
#include "radix_sort.h"
#define BITS 4

void radix_sort(Data *data) {

    if (data->array_used == INT) {
        radix_sort_int(data);
    } else if (data->array_used == FLOAT) {
        radix_sort_float(data);
    }

};

long long find_max_int (Data *data) {
        unsigned long long length = data->length;
        int *a = data->intarray;
        long long max = a[0];

        for (unsigned long long i = 1; i < length; i++) {
                if (a[i] > max) {
                        max = a[i];
                }
        }

        return max;


}

void radix_sort_int(Data *data) {

        long long max = find_max_int(data);
        unsigned long long length = data->length;
        int *a = data->intarray;

        int b[length];

        int buckets = 1 << BITS;
        int mask = buckets - 1;

        int pos = 0;

        while (max >> (pos * BITS) > 0) {
                int bucket[1 << BITS] = {0};

                for (int i = 0; i < length; i++) {
                        bucket[(a[i] >> (pos * BITS)) & mask]++;
                }

                for (int i = 1; i < buckets; i++) {
                        bucket[i] += bucket [i - 1];
                }

                for (int i = length - 1; i >= 0; i--) {
                        b[--bucket[(a[i] >> (pos * BITS)) & mask]] = a[i];
                }

                for (int i = 0; i < length; i++) {
                        a[i] = b[i];
                }

                pos++;
        }


}

void radix_sort_float(Data *data) {



}
