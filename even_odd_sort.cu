#include "hip/hip_runtime.h"

#include "even_odd_sort.h"
 
 
 
 
__global__ void even_sort(int* ary, int size)
{
	int tid = (blockIdx.z * gridDim.x * gridDim.y
                + blockIdx.y * gridDim.x
                + blockIdx.x) * blockDim.x + threadIdx.x;
	if(1 == (tid + 1 ) % 2) 
	{
		 
		if(ary[tid] > ary[tid + 1] && tid + 1 < size)
		{
			int tp = ary[tid];
			ary[tid] = ary[tid + 1];
			ary[tid + 1] = tp;
 
		}
	}
	__syncthreads();
 
}
__global__ void odd_sort(int* ary, int size)
{
	int tid = (blockIdx.z * gridDim.x * gridDim.y
                + blockIdx.y * gridDim.x
                + blockIdx.x) * blockDim.x + threadIdx.x;
	if(0 == (tid + 1 ) % 2) 
	{
 
		if(ary[tid] > ary[tid + 1] && tid + 1 < size)
		{
			int tp = ary[tid];
			ary[tid] = ary[tid + 1];
			ary[tid + 1] = tp;
 
		}
	}
	__syncthreads();
}
void even_odd_sort(Data *data)
{
	const int SIZE = data->length;
  
    int threads;
    Grid grid;
    int size = 1;
	cal_grid(&grid, &threads, data->length, size);	 
    dim3 blocks(grid.blockx, grid.blocky, grid.blockz);
    int* gary;
	hipMalloc((void**) &gary, SIZE * sizeof(int));
	hipMemcpy(gary, data->intarray, SIZE * sizeof(int), hipMemcpyHostToDevice);
	for(int i = 0; i < SIZE; ++ i)
	{
	    if (i % 2 == 0) {
		    even_sort<<<blocks, threads>>>(gary, SIZE);
		} else {
		    odd_sort<<<blocks, threads>>>(gary, SIZE);
		}
	}
	 
	hipMemcpy(data->intarray, gary, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	 
	hipFree(gary);
	 
	 
}
