#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>
#include <errno.h>
#include <time.h>
#include "structs.h"
#include "array_maker.h"
#include "insertion.h"
#include "mergeSort.h"

typedef long long Parses(const char *str, char **endptr, int base);
typedef unsigned long long Parseu(const char *str, char **endptr, int base);

long long parse_signed(char *arg, char *str, Parses *parseint);
unsigned long long parse_unsigned(char *arg, char *str, Parseu *parseuint);


/* Parse command line arguments, generate a random array to the client's 
 * specifications, and pass the array to the specified sorting algorithms */
int main (int argc, char *argv[])
{

        Data_info *specs = (Data_info *)malloc(sizeof(*specs));
        specs->sample_size = 10000;
        specs->max = 100000;
        specs->min = -10;

        Sort algorithm;
        Concurrency parallelism;

        /* ===========================================

                    Main loop though CL args

           =========================================== */
        for (int i = 1; i < argc; i++) {
                char *arg = argv[i];

                /* ========================================
                       -n, -num, and -number arguments 
                   ======================================== */
                if (strcmp(arg, "-n") == 0 || strcmp(arg, "-num") == 0
                                      || strcmp(arg, "-number") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "a value.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *num_str = argv[i];

                        unsigned long long num = 
                                       parse_unsigned(arg, num_str, &strtoull);

                        specs->sample_size = num;
                }

                /* ========================================
                    -d, -dist, and -distribution arguments 
                   ======================================== */
                else if (strcmp(arg, "-d") == 0 || strcmp(arg, "-dist") == 0
                                      || strcmp(arg, "-distribution") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "a value.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *dist_str = argv[i];

                        if (strcmp(dist_str, "gaussian") == 0 || 
                            strcmp(dist_str, "g") == 0) {
                                specs->dist = GAUSSIAN;

                        } else if (strcmp(dist_str, "uniform") == 0 || 
                                   strcmp(dist_str, "u") == 0) {
                                specs->dist = UNIFORM;

                        } else if (strcmp(dist_str, "constant") == 0 || 
                                   strcmp(dist_str, "c") == 0) {
                                specs->dist = CONSTANT;

                        } else if (strcmp(dist_str, "quadratic") == 0 || 
                                   strcmp(dist_str, "q") == 0) {
                                specs->dist = QUADRATIC;

                        } else if (strcmp(dist_str, "buckets") == 0 || 
                                   strcmp(dist_str, "b") == 0) {
                                specs->dist = BUCKETS;

                        } else {
                                fprintf(stderr, "ERROR: the distribution "
                                                "specified (%s) was not "
                                                "recognized.\n", dist_str);
                                exit(EXIT_FAILURE);
                        }
                      
                /* ========================================
                           -t and -type arguments 
                   ======================================== */  
                } else if (strcmp(arg, "-t") == 0 || 
                           strcmp(arg, "-type") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "a value.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *type_str = argv[i];

                        if (strcmp(type_str, "int") == 0 || 
                            strcmp(type_str, "i") == 0) {
                                specs->contents = INT;

                        } else if (strcmp(type_str, "fp") == 0 || 
                                   strcmp(type_str, "f") == 0 || 
                                   strcmp(type_str, "float") == 0 || 
                                   strcmp(type_str, "floating-point") == 0) {
                                specs->contents = FLOAT;
                        } else {
                                fprintf(stderr, "ERROR: the datatype "
                                                "specified (%s) was not "
                                                "recognized.\n", type_str);
                                exit(EXIT_FAILURE);
                        }

                /* ========================================
                          -r and -range arguments 
                   ======================================== */
                } else if (strcmp(arg, "-r") == 0 ||
                           strcmp(arg, "-range") == 0) {
                      
                        if (i + 2 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "enough values.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *min_str = argv[i];
                        i++;
                        char *max_str = argv[i];

                        long long min = parse_signed(arg, min_str, &strtoll);
                        long long max = parse_signed(arg, max_str, &strtoll);

                        specs->min = min;
                        specs->max = max;

                /* ========================================
                           -s and -sort arguments 
                   ======================================== */

                } else if (strcmp(arg, "-s") == 0 ||
                           strcmp(arg, "-sort") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "enough values.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *sort_name = argv[i];

                        if (strcmp(sort_name, "quick") == 0 ||
                            strcmp(sort_name, "quicksort") == 0) {
                                algorithm = QUICK;

                        } else if (strcmp(sort_name, "shell") == 0 ||
                            strcmp(sort_name, "shell-sort") == 0) {
                                algorithm = SHELL;

                        } else if (strcmp(sort_name, "merge") == 0||
                            strcmp(sort_name, "merge-sort") == 0) {
                                algorithm = MERGE;

                        } else if (strcmp(sort_name, "insertion") == 0 ||
                            strcmp(sort_name, "insertion-sort") == 0) {
                                algorithm = INSERTION;

                        } else if (strcmp(sort_name, "heap") == 0 ||
                            strcmp(sort_name, "heap-sort") == 0) {
                                algorithm = HEAP;

                        } else if (strcmp(sort_name, "radix") == 0 ||
                            strcmp(sort_name, "radix-sort") == 0) {
                                algorithm = RADIX;

                        } else if (strcmp(sort_name, "bitonic") == 0 ||
                            strcmp(sort_name, "bitonic-sort") == 0) {
                                algorithm = BITONIC;

                        } else {
                                fprintf(stderr, "No valid algorithm specified"
                                                " after %s", arg);
                                exit(EXIT_FAILURE);
                        }

                /* ========================================
                           -p and -parallel arguments 
                   ======================================== */

                } else if (strcmp(arg, "-p") == 0 ||
                           strcmp(arg, "-parallel") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "enough values.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *par = argv[i];

                        if (strcmp(par, "p") == 0 ||
                            strcmp(par, "parallel") == 0) {
                                parallelism = PARALLEL;

                        } else if (strcmp(par, "s") == 0 ||
                            strcmp(par, "serial") == 0) {
                                parallelism = SERIAL;

                        } else if (strcmp(par, "b") == 0 ||
                            strcmp(par, "both") == 0) {
                                parallelism = BOTH;

                        } else {
                                fprintf(stderr, "No valid parallelism "
                                                "specified after %s", arg);
                                exit(EXIT_FAILURE);
                        }

                 /* ========================================
                           -o and -order arguments 
                   ======================================== */

                } else if (strcmp(arg, "-o") == 0 ||
                           strcmp(arg, "-order") == 0) {
                      
                        if (i + 1 == argc) {
                                fprintf(stderr, "ERROR: command line argument "
                                                "%s was not followed by "
                                                "enough values.\n", arg);
                                exit(EXIT_FAILURE);
                        }

                        i++;
                        char *order = argv[i];
 
                        if (strcmp(order, "s") == 0 ||
                            strcmp(order, "sorted") == 0) {
              
                                specs->order = SORTED;

                        } else if (strcmp(order, "r") == 0 ||
                            strcmp(order, "reversed") == 0) {
                                specs->order = REVERSED;

                        } else if (strcmp(order, "n") == 0 ||
                            strcmp(order, "nearly-sorted") == 0) {
                                specs->order = NEARLYSORTED;

                        } else if (strcmp(order, "u") == 0 ||
                            strcmp(order, "unsorted") == 0 ||
                            strcmp(order, "random") == 0 ||
                            strcmp(order, "shuffled") == 0) {
                                specs->order = UNSORTED;

                        }else {
                                fprintf(stderr, "No valid array sortedness "
                                                "specified after %s", arg);
                                exit(EXIT_FAILURE);
                        }
                }

               
        }

        fprintf(stderr, "Parsed command-line arguments.\n\n");

        fprintf(stderr, "Will create & sort array with properties:\n");

        fprintf(stderr, "Length: %llu\n", specs->sample_size);

        if (specs->dist == UNIFORM) {
                fprintf(stderr, "Distribution: uniform\n");
        } else if (specs->dist == GAUSSIAN) {
                fprintf(stderr, "Distribution: gaussian\n");
        } else if (specs->dist == CONSTANT) {
                fprintf(stderr, "Distribution: constant\n");
        } else if (specs->dist == QUADRATIC) {
                 fprintf(stderr, "Distribution: quadratic\n");
        } else if (specs->dist == BUCKETS) {
                fprintf(stderr, "Distribution: buckets\n");
        }

        if (specs->contents == INT) {
                fprintf(stderr, "Elements of type: int\n");
        } else if (specs->contents == FLOAT) {
                fprintf(stderr, "Elements of type: float\n");
        }

        fprintf(stderr, "Minimum element: %lld\n", specs->min);
        fprintf(stderr, "Maximum element: %lld\n", specs->max);

        if (specs->order == SORTED) {
                fprintf(stderr, "Sortedness: sorted\n\n");
        } else if (specs->order == REVERSED) {
                fprintf(stderr, "Sortedness: reversed\n\n");
        } else if (specs->order == NEARLYSORTED) {
                fprintf(stderr, "Sortedness: nearly sorted\n\n");
        } else if (specs->order == UNSORTED) {
                fprintf(stderr, "Sortedness: random\n\n");
        } 

        if (algorithm == ALL) {
                fprintf(stderr, "With all algorithms\n");
        } else if (algorithm == BITONIC) {
                fprintf(stderr, "With algorithm: bitonic sort\n");
        } else if (algorithm == RADIX) {
                fprintf(stderr, "With algorithm: radix sort\n");
        } else if (algorithm == QUICK) {
                fprintf(stderr, "With algorithm: quicksort\n");
        } else if (algorithm == MERGE) {
                fprintf(stderr, "With algorithm: merge sort\n");
        } else if (algorithm == SHELL) {
                fprintf(stderr, "With algorithm: shellsort\n");
        } else if (algorithm == HEAP) {
                fprintf(stderr, "With algorithm: heap sort\n");
        } else if (algorithm == INSERTION) {
                fprintf(stderr, "With algorithm: insertion sort\n");
        } 

        if (parallelism == BOTH) {
                fprintf(stderr, "Running both in serial "
                                "and parallel\n\n");
        } else if (parallelism == SERIAL) {
                fprintf(stderr, "Running in serial\n\n");
        } else if (parallelism == PARALLEL) {
                fprintf(stderr, "Running in parallel\n\n");
        }

        srand(time(NULL));
        Data *data = generate_data(specs);
        print_array(data, specs);
        if (algorithm == ALL) {
                fprintf(stderr, "With all algorithms, to be done\n");
        } else if (algorithm == BITONIC) {
                fprintf(stderr, "With algorithm: bitonic sort to be done\n");
        } else if (algorithm == RADIX) {
                fprintf(stderr, "With algorithm: radix sort to be done\n");
        } else if (algorithm == QUICK) {
                fprintf(stderr, "With algorithm: quicksort to be done\n");
        } else if (algorithm == MERGE) {
                fprintf(stderr, "Invoking algorithm: merge sort\n");
                mergeSort(data);
        } else if (algorithm == SHELL) {
                fprintf(stderr, "With algorithm: shellsort to be done\n");
        } else if (algorithm == HEAP) {
                fprintf(stderr, "With algorithm: heap sort to be done\n");
        } else if (algorithm == INSERTION) {
                fprintf(stderr, "Invoking algorithm: insertion sort\n");
                insertion(data);
        } 


}

long long parse_signed(char *arg, char *str, Parses *parseint) {
        
        int digits = strlen(str);

        if (digits <= 0) {
                fprintf(stderr, "ERROR: Impossibly, an arg"
                               " was an empty string.\n");
                exit(EXIT_FAILURE);

        }

        long long num = parseint(str, NULL, 0);

        if (errno == ERANGE || errno == EINVAL) {
                fprintf(stderr, "ERROR: command line argument"
                                " %s was not followed by a "
                                "valid value. Value provided "
                                "was: %s.\n", arg, str);
                exit(EXIT_FAILURE);
        } else if (num == 0 && digits != 1 && str[0] == '0') {
                fprintf(stderr, "ERROR: the value following "
                                "argument %s was not able to "
                                "be interpreted as an integer."
                                " Value was: %s\n", arg, 
                                str);
                exit(EXIT_FAILURE);
        }

        return num;
}

unsigned long long parse_unsigned(char *arg, char *str, Parseu *parseuint) {
        
        int digits = strlen(str);

        if (digits <= 0) {
                fprintf(stderr, "ERROR: Impossibly, an arg"
                               " was an empty string.\n");
                exit(EXIT_FAILURE);

        } else if (str[0] == '-') {
                fprintf(stderr, "ERROR: value passed to argument %s cannot "
                                "be a negative number. Value "
                                "was %s.\n", arg, str);
                exit(EXIT_FAILURE);
        }

        unsigned long long num = 
                        parseuint(str, NULL, 0);

        if (errno == ERANGE || errno == EINVAL) {
                fprintf(stderr, "ERROR: command line argument"
                                " %s was not followed by a "
                                "valid value. Value provided "
                                "was: %s.\n", arg, str);
                exit(EXIT_FAILURE);
        } else if (num == 0 && digits != 1 && str[0] == '0') {
                fprintf(stderr, "ERROR: the value following "
                                "argument %s was not able to "
                                "be interpreted as an integer."
                                " Value was: %s\n", arg, 
                                str);
                exit(EXIT_FAILURE);
        }

        return num;
}
