#include "hip/hip_runtime.h"

#include "parallel_radix_sort.h"

void parallel_radix_sort(Data *data) {

        if (data->array_used == INT) {
                parallel_radix_sort_int(data);
        } else if (data->array_used == FLOAT) {
                parallel_radix_sort_float(data);
        }

};

void parallel_radix_sort_int(Data *data)
{

        clock_t begin, end;
        double time_spent;

        int *a = data->intarray;
        thrust::host_vector<int> h_a;
        thrust::device_vector<int> d_a;

        for (int i = 0; i < data->length; i++) {
                h_a.push_back(a[i]);
        }

        begin = clock();

        d_a = h_a;
        thrust::sort(d_a.begin(), d_a.end());
        h_a = d_a;

        end = clock();

        for (int i = 0; i < data-> length; i++) {
                a[i] = h_a[i];
        }

        time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

        fprintf(stdout, "Parallel Radix time: %f\n", time_spent);

}

void parallel_radix_sort_float(Data *data)
{

}

