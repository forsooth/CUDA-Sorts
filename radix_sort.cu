#include "hip/hip_runtime.h"
#include "radix_sort.h"
#ifndef BITS
#define BITS 9
#endif

void radix_sort(Data *data) {

    if (data->array_used == INT) {
        radix_sort_int(data);
    } else if (data->array_used == FLOAT) {
        radix_sort_float(data);
    }

};

long long *find_max_int (Data *data) {
        unsigned long long length = data->length;
        int *a = data->intarray;
        long long max = a[0];
        long long min = a[0];

        for (unsigned long long i = 1; i < length; i++) {
                if (abs(a[i]) > max) {
                        max = abs(a[i]);
                }
                if (a[i] < min) {
                        min = a[i];
                }
        }

        long long *maxmin = (long long *) malloc(2 * sizeof(*maxmin));
        maxmin[0] = max;
        maxmin[1] = min;

        return maxmin;
}

void radix_sort_int(Data *data) {

        long long *maxmin = find_max_int(data);
        long long max = maxmin[0];
        long long min = maxmin[1];
        free(maxmin);
        unsigned long long length = data->length;
        int *a = data->intarray;

        int *b = (int *) malloc(length * sizeof(*b));

        int buckets = 1 << BITS;
        int mask = buckets - 1;

        int pos = 0;

        while (max >> (pos * BITS) > 0) {
                int bucket[1 << BITS] = {0};

                for (int i = 0; i < length; i++) {
                        bucket[(a[i] >> (pos * BITS)) & mask]++;
                }

                for (int i = 1; i < buckets; i++) {
                        bucket[i] += bucket[i - 1];
                }

                for (int i = length - 1; i >= 0; i--) {
                        b[--bucket[(a[i] >> (pos * BITS)) & mask]] = a[i];
                }

                for (int i = 0; i < length; i++) {
                        a[i] = b[i];
                }

                pos++;
        }

        if (min < 0) {

                int bucket[2] = {0};

                for (int i = 0; i < length; i++) {
                        bucket[(a[i] < 0)]++;
                }

                bucket[0] += bucket[1];

                for (int i = length - 1; i >= 0; i--) {
                        b[--bucket[(a[i] < 0)]] = a[i];
                }

                for (int i = 0; i < length; i++) {
                        a[i] = b[i];
                }
        }

        free(b);


}

void radix_sort_float(Data *data) {



}
