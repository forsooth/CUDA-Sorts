#include "hip/hip_runtime.h"
#include "shellsort.h"


void shellsort(Data *data) {

        if (data->array_used == INT) {
                shellsort_int(data);
        } else if (data->array_used == FLOAT) {
                shellsort_float(data);
        }

};

unsigned long long *generate_gaps(unsigned long long length) {

        int num_gaps = log(length) / log(2.48);

        unsigned long long *gaps = (unsigned long long *) malloc((num_gaps + 1) * sizeof(*gaps));

        for (int i = 0; i <= num_gaps; i++) {
                gaps[i] = pow(2.48, i);
                #ifdef PRINTGAPS
                fprintf(stderr, "%llu, ", gaps[i]);
                #endif
        }
        #ifdef PRINTGAPS
        fprintf(stderr, "\n");
        #endif

        return gaps;
}


void shellsort_int(Data *data) {
	clock_t begin, end;
        double time_spent;
        begin = clock();

        int* a = data->intarray;
        unsigned long long length = data->length;

        unsigned long long *gaps = generate_gaps(length);
        int num_gaps = sizeof(gaps) / sizeof(gaps[0]);

        int gapi = num_gaps;
        unsigned long long gap;
        int temp, j, i;

        while (gapi >= 0) {
                gap = gaps[gapi];
                for (i = gap; i < length; i++) {
                        temp = a[i];
                        for (j = i; (j >= gap) && (a[j - gap] > temp); j -= gap) {
                                a[j] = a[j - gap];
                        }

                        a[j] = temp;
                }

                gapi--;
        }

        free(gaps);

	end = clock();
        time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        fprintf(stdout, "Serial Shellsort time: %f\n", time_spent);


}

void shellsort_float(Data *data) {



}

