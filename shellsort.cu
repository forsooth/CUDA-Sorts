#include "hip/hip_runtime.h"
#include "shellsort.h"


void shellsort(Data *data) {

        if (data->array_used == INT) {
                shellsort_int(data);
        } else if (data->array_used == FLOAT) {
                shellsort_float(data);
        }

};

unsigned long long *generate_gaps(unsigned long long length) {

        int num_gaps = log(length) / log(2.48);

        unsigned long long *gaps = (unsigned long long *) malloc((num_gaps + 1) * sizeof(*gaps));

        for (int i = 0; i <= num_gaps; i++) {
                gaps[i] = pow(2.48, i);
                #ifdef PRINTGAPS
                fprintf(stderr, "%llu, ", gaps[i]);
                #endif
        }
        #ifdef PRINTGAPS
        fprintf(stderr, "\n");
        #endif

        return gaps;
}


void shellsort_int(Data *data) {

        int* a = data->intarray;
        unsigned long long length = data->length;

        unsigned long long *gaps = generate_gaps(length);
        int num_gaps = sizeof(gaps) / sizeof(gaps[0]);

        int gapi = num_gaps;
        unsigned long long gap;
        int temp, j, i;

        while (gapi >= 0) {
                gap = gaps[gapi];
                for (i = gap; i < length; i++) {
                        temp = a[i];
                        for (j = i; (j >= gap) && (a[j - gap] > temp); j -= gap) {
                                a[j] = a[j - gap];
                        }

                        a[j] = temp;
                }

                gapi--;
        }

        free(gaps);

}

void shellsort_float(Data *data) {



}

