#include "hip/hip_runtime.h"
#include"quicksort_parallel.h"
#include<stdio.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        int min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            int val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void quicksort_parallel_int(int *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    int *lptr = data+left;
    int *rptr = data+right;
    int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        int lval = *lptr;
        int rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort_parallel_int<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_parallel_int<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}


void quicksort_parallel(Data* data) {

       if (data->array_used == INT) {
	        clock_t begin, end;
	        double time_spent;
	        begin = clock();

    		int left = 0;
		int right = data->length-1;
		printf("Launching kernel on the GPU");
		quicksort_parallel_int<<< 1, 1 >>>(data->intarray, left, right, 0);
		checkCudaErrors(hipDeviceSynchronize());
	        end = clock();
	        time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	        fprintf(stdout, "Parallel Quicksort time: %f\n", time_spent);

        } else if (data->array_used == FLOAT) {//TODO: Add float support
	      //  clock_t begin, end;
	      //  double time_spent;
	      // begin = clock();

    		//int left = 0;
		//int right = data->length-1;
		//printf("Launching kernel on the GPU");
		//quicksort_parallel_float<<< 1, 1 >>>(data->floatarray, left, right, 0);
		//checkCudaErrors(hipDeviceSynchronize());
	        //end = clock();
	        //time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	        //fprintf(stdout, "Parallel Quicksort time: %f\n", time_spent);
        }


    // Launch on device

    


	return 0;
}

